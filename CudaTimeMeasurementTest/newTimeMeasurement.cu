#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <string>
#include <vector>
#include <random>
#include <algorithm>
#include <utility>
#include <chrono>  //Time
#include<tuple>
#include <cassert>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <unistd.h>
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "newTimeCalc.h"
#include "Timing.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

std::tuple<int, std::vector<int>, int> parseCommands(int argc, char* argv[]);
void printResults(int vecSize, int iterations, Timing time, int commMethods);
void saveToFile(const std::string &name, const Timing &timing, int vecSize, int iterations);
void printHeader();

int main(int argc, char* argv[]) {
    cms::cudatest::requireDevices(); //pre-built test 

    auto [vecSize, commMethods, iterations] = parseCommands(argc,argv);
    
    std::vector<Timing> results; 
    std::cout << "\n";
    calculate calc(iterations, vecSize);

    for (size_t i = 0; i < commMethods.size(); i++) {
            std::cout << "comm methods: " << commMethods[i] << std::endl;
    }
    for (size_t i = 0; i < commMethods.size(); i++) {
    std::cout << "comm methods: " << commMethods[i] << std::endl;
	results.push_back(calc.calculateAverage(commMethods[i]));
  }
    printHeader();
    for (size_t i = 0; i < commMethods.size(); i++){
        printResults(vecSize, iterations ,results[i], commMethods[i]);
    }
    return 0;
}

void printHeader(){
/*    std::cout << "error: " << time.noError <<
                "IN MAIN Average upload: " << time.timeUploadAvg <<
                " std upload: " << time.timeUploadstd <<
                " Average download: " << time.timeDownloadAvg <<
                " std download: " << time.timeDownloadstd <<
                " Average calc dev: " << time.timeCalcAvg <<
                " std calc dev: " << time.timeCalcstd <<
                " average calc host: " << time.timeCalcCpuAvg <<
                " std calc host: " << time.timeCalcCpustd ;
*/
    const auto COL1 = 25, COL2to10 = 15, COL11 = 11;
    std::string ROW    = "=====================================================================================================";
    std::string DASHES = "-----------------------------------------------------------------------------------------------------";
    std::cout.flags(std::ios::fixed | std::ios::showpoint);
    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(6);

    std::cout << "\n\n\t"<<ROW<<ROW;
    std::cout << "\n\t|| "<<std::left
              <<std::setw(COL1)<<"Communication Method"<<"|| "
              <<std::setw(COL2to10)<<"Upload Avg"<<"|| "
              <<std::setw(COL2to10)<<"Upload Std"<<"|| "
              <<std::setw(COL2to10)<<"Calc Avg"<<"|| "
              <<std::setw(COL2to10)<<"Calc Std"<<"|| "
              <<std::setw(COL2to10)<<"Download Avg"<<"|| "
              <<std::setw(COL2to10)<<"Download Std"<<"|| "
              <<std::setw(COL2to10)<<"Cpu pov Avg"<<"|| "
              <<std::setw(COL2to10)<<"cpu pov Std"<<"|| "
              <<std::setw(COL11)<<"Iterations"<<"||"
              <<std::setw(COL11)<<"Error?"<<"||"
              << "\n\t"<<ROW <<ROW;
}

void printResults(int vecSize, int iterations, Timing time, int commMethods) {
    const std::string COMM_METHOD_NAMES[] = {"Method0","Method1","Method2" ,"Method3", "Method4", "Method5"};
 
    const auto COL1 = 25, COL2to10 = 15, COL11 = 11;
    std::string ROW    = "=====================================================================================================";
    std::string DASHES = "-----------------------------------------------------------------------------------------------------";
    std::cout.flags(std::ios::fixed | std::ios::showpoint);
    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(6);

    //std::cout << "\n\t"<<DASHES<<DASHES;   
    std::string error;
    if(!time.noError){
       error = "Error";
    }else{
       error = "no Error";
        }
              
    std::cout << "\n\t|| " <<std::left
                <<std::setw(COL1) << COMM_METHOD_NAMES[commMethods - 1] << "|| "
                <<std::setw(COL2to10)<<time.timeUploadAvg<<"|| "
                <<std::setw(COL2to10)<<time.timeUploadstd<<"|| "
                <<std::setw(COL2to10)<<time.timeCalcAvg<<"|| "
                <<std::setw(COL2to10)<<time.timeCalcstd<<"|| "
                <<std::setw(COL2to10)<<time.timeDownloadAvg<<"|| "
                <<std::setw(COL2to10)<<time.timeDownloadstd<<"|| "
                <<std::setw(COL2to10)<<time.timeCalcCpuAvg<<"|| "
                <<std::setw(COL2to10)<<time.timeCalcCpustd<<"|| "
                <<std::setw(COL11) << iterations<< "||"
                <<std::setw(COL11) << error << "||";
	     
    std::cout << "1\n\t"<<ROW<<ROW;
    saveToFile(COMM_METHOD_NAMES[commMethods - 1], time , vecSize, iterations);
    
}

void saveToFile(const std::string &name, const Timing &timing, int vecSize, int iterations) {
  std::ofstream file(name + ".txt", std::ios::out | std::ios::app);

  if (!file.is_open()) {
    std::cout << "\nCannot open File nor Create File!" << std::endl;
  }
  file << "| Vector Size: " << vecSize << " |" << "\n| Number of iterations : " << vecSize << " |" << std::endl;
  file << "upload to device: \n" << timing.timeUploadAvg << " " << timing.timeUploadstd << std::endl;
  file << "calculate in kernel: \n" << timing.timeCalcAvg << " " << timing.timeCalcstd << std::endl;
  file << "upload to host: \n" <<  timing.timeDownloadAvg << " " << timing.timeDownloadstd << std::endl;
  file << "calculate in kernel - CPU POV: \n" <<  timing.timeCalcCpuAvg << " " << timing.timeCalcCpustd << std::endl;
  file << "----------------------------------------------------------------" << std::endl;
  file.close();
  if (!file.good()) {
    std::cout << "\n*ERROR While Writing The " + name + " file!!" << std::endl;
  }
}

std::tuple<int, std::vector<int>, int> parseCommands(int argc, char* argv[]){
    const int METHODS_COUNT = 6; 
    enum INPUT_OPTIONS { VECTOR_SIZE = 's', COMMUNICATION_METHOD = 'f', ITERATIONS = 'i'};
    
    //default values
    int vecSize = 100; 
    int iterations = 10; 
    std::vector<int> commMethods; 

    int input;   // Parsing command-line arguments
   
    while ((input = getopt(argc, argv, "s:f:i:")) != -1) {              
        switch (input) {
		case VECTOR_SIZE:
                try {
                    vecSize = std::stoll(optarg, nullptr, 0);
                } catch (std::exception& err) {
                    std::cout << "\n\tError: Argument s must be an integer!";
                    std::cout << "\n\t" << err.what() << std::endl;
                    abort();
                }
                break;
		case COMMUNICATION_METHOD:
                try {
                  int inputNum = std::stoll(optarg, nullptr, 0); // Sending Methods selected by user (e.g. 34 user selected methods blocking and nonblocking)
                  std::cout << "this is the input: " << inputNum;
	    	    while(inputNum > 0 ){
			    int digit = inputNum % 10; 
			    if (digit > METHODS_COUNT) {
				    //FIXME: Raise an exception here
				    std::cout << "\n\tError: Argument must be an integer <= " << METHODS_COUNT << std::endl;
				    abort(); 
			    }
			    commMethods.push_back(digit); 
			    inputNum = inputNum / 10; 
		    }
                } catch (std::exception& err) {
                    std::cout << "\n\tError: Argument r must be an integer!";
                    std::cout << "\n\t" << err.what() << std::endl;
                    abort();
                }
                break;
		case ITERATIONS:
                try {
                    iterations = std::stoll(optarg, nullptr, 0);  // Set the average run count based on the command-line argument.
                } catch (std::exception& err) {
                    std::cout << "\n\tError: Argument n must be an integer!";
                    std::cout << "\n\t" << err.what() << std::endl;
                    abort();
                }
                break;
            default:
		std::cerr<<"\n\t WRONGE INPUT ****** ABORT! "<<input<<"\n"; 
                abort();
        }
    }
    return std::make_tuple(vecSize, commMethods, iterations); 
}

